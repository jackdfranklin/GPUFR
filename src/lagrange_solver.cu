#include "hip/hip_runtime.h"
#include "GPUFR/lagrange_solver.cuh"

// TODO: Make this dynamic
#define MAX_VARS 6 // The maximum number of variable to reconstruct over
#define MAX_EXPONENT 101 // The maximum exponent in the polynomeal
#define UNSIGNED_TYPE unsigned
#define PRIME 105097513 // Must be less than (max unsigend) / 2

__host__ __device__ int as_int(u32 val)
{
    int result = val;
    if (result > PRIME/2) result = result - PRIME;
    return result;
}

__host__ __device__ void print_vec(const u32* vec, int size)
{
    for (int i=0; i<size; i++)
    {
        printf("%i, ", as_int(vec[i]));
    }
    printf("\n");
}

__device__ u32 fun(u32 *vars)
{
    u32 result;
    result = vars[0];
    return result;
}

__global__ void compute_probes(const u32 *xs, u32 *probes, u32 *probes_2, int n_vars, int n_samps) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    u32 test_params[MAX_VARS];

    for (int j=0; j<n_vars; j++)
    {
        float ifloat = i;
        int dimension_index = static_cast<int>(floorf(ifloat/pow(n_samps, j))) % n_samps;//floorf((pow(n_samps, j)));
        test_params[j] = xs[j*n_samps+dimension_index];
    }
        
    probes[i] = fun(test_params);
    probes_2[i] = 0;
}


__device__ void atomic_add(u32 *l_val, u32 r_val)
{
    u32 assumed, old;
    u32 value;

    old = (*l_val);
    do
    {
        value = *l_val;
        assumed = old;
        old = atomicCAS(&((*l_val)), assumed, ff_add(value, r_val, PRIME));
    } while (assumed != old);
}

__device__ u32 compute_denom_nd(int current_index, const u32 *xs, int dim, int n_vars, int n_samps, int idx)
{
    int flat_current_index = dim*n_samps + static_cast<int>(floorf(idx/pow(n_samps, dim)))%n_samps;

    u32 denom;
    denom = 1;

    for (int i=0; i<n_samps; i++)
    {
        int flat_index = dim*n_samps + i;
        if (flat_index != flat_current_index)
        {
            denom = ff_multiply(denom, (ff_subtract(xs[flat_current_index],  xs[flat_index], PRIME)), PRIME);
        }
    }

    return denom;
}

__global__ void get_lagrange_coeffs_nd(const u32 *xs, u32 *ys, u32 *out, const u32 *lagrange, int dim, int n_vars, int n_samps, int two_exponent)
{
    // Computes the coefficients to the Lagrange polynomials and writes them to ys

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int pol_size = pow(2, two_exponent) + 1;
    int step_size = 2*(n_samps-1) - pol_size;

    // if (idx == 1){
    // printf("pol_size: %i \n", pol_size);
    // printf("step_size: %i \n", step_size);
    // }

    u32 denom;
    denom = compute_denom_nd(idx, xs, dim, n_vars, n_samps, idx);

    u32 coefficient = ff_divide(ys[idx], denom, PRIME);

    int index_step_ys = pow(n_samps, dim);
    int power = static_cast<int>(floorf(pow(n_samps, dim+1)));
    int start_index_ys = floorf(idx / power)*power + idx%index_step_ys;

    int index_xs = dim*n_samps + static_cast<int>(floorf(idx/pow(n_samps, dim)))%n_samps;

    for (int i=0; i<n_samps; i++)
    {
        int flat_index_ys = start_index_ys+i*index_step_ys;
        int flat_index_lagrange = index_xs*n_samps + i;
        int actual_index_lagrange = step_size*(flat_index_lagrange/pol_size) + flat_index_lagrange;
        // printf("idx: %i i: %i probe index: %i start index: %i to add: %i denom: %i lag: %i val: %i y: %i \n", idx, i, flat_index_ys, power, as_int(coefficient), as_int(denom), as_int(lagrange[flat_index_lagrange]), as_int(ff_multiply(ff_divide(lagrange[flat_index_lagrange], denom, PRIME), ys[idx], PRIME)), ys[idx]);
        u32 to_add = ff_multiply(ff_divide(lagrange[actual_index_lagrange], denom, PRIME), ys[idx], PRIME);
        atomic_add(&out[flat_index_ys], to_add); // TODO: more efficient reduction, this is the current bottleneck
    }

    ys[idx] = 0;

}


void convolve_cpp(const u32 *kernel, const u32 *signal, u32 *out, int kernel_size, int signal_size)
{
    int result_size = signal_size+kernel_size-1;
    int pad_size = kernel_size-1;
    for (int i=0; i<result_size; i++)
    {
        out[i] = 0;
        for (int j = 0; j < kernel_size; j++)
        {
            if (i+j >= pad_size && i+j-pad_size < signal_size)
            {
                out[i] = ff_add(out[i], ff_multiply(kernel[kernel_size - 1 - j], signal[i+j-pad_size], PRIME), PRIME);
            }
        }
    }
}

void compute_lagrange_pol(const u32 *xs, u32 *lagrange, int dim, int n_vars, int n_samps)
{
    u32 root_arr[2];
    u32 tmp[MAX_EXPONENT];
    u32 tmp2[MAX_EXPONENT];

    // Loop over each x to get l(x, xi)
    for (int i=0; i<n_samps; i++)
    {
        tmp[0] = 1.0;
        for (int k=1; k<n_samps; k++)
        {
            tmp[k] = 0.0;
        }


        // Iteratively convolve to compute expansion
        for (int j=0; j<n_samps; j++)
        {
            int x_index = dim*n_samps + j;

            if (i != j)
            {
                root_arr[0] = ff_subtract(0, xs[x_index], PRIME);
                root_arr[1] = 1;

                convolve_cpp(tmp, root_arr, tmp2, n_samps, 2);

                for (int k=0; k<n_samps; k++)
                {
                    tmp[k] = tmp2[k];
                }
            }
        }

        // Copy currnet expansion into lagrange
        for (int j=0; j<n_samps; j++)
        {
            int x_index = dim*n_samps + i;
            int lagrange_index = x_index*n_samps + j;

            lagrange[lagrange_index] = tmp[j];

            int pLag = tmp[j];
            if (pLag > PRIME/2) pLag = tmp[j] - PRIME;
            // printf("idx: %i term: %i expansion: %i \n", i, j, pLag);
        }
    }
}


std::string nd_poly_to_string_flat(const std::vector<double>& coef_flat, const std::vector<std::string>& variables, int n_samps) {
    // From chat GPT
    int dim = variables.size();
    std::ostringstream result;
    for (size_t i = 0; i < coef_flat.size(); ++i) {
        double c = coef_flat[i];
        if (sqrt(pow(c, 2)) >= 1) {
            c = c > PRIME/2.0 ? c-PRIME : c;
            result << (c > 0 && result.tellp() > 0 ? "+ " : "") << std::fixed << std::setprecision(0) << c;
            for (int j = 0; j < dim; ++j) {
                int power = static_cast<int>(std::floor(i / std::pow(n_samps, j))) % n_samps;
                if (power > 0) {
                    result << "*" << variables[j] << "^" << power;
                }
            }
            result << " ";
        }
    }
    return result.str();
}

// __device__ void convolve_gpu(u32* kernel, u32* signal, u32* out, int out_start_loc, int in_size)
// {
//     int result_size = in_size+in_size-1;
//     int pad_size = in_size-1;
//     for (int i=0; i<result_size; i++)
//     {
//         int out_indx = out_start_loc+i;
//         out[out_indx] = 0;
//         for (int j = 0; j < in_size; j++)
//         {
//             if (i+j >= pad_size && i+j-pad_size < in_size)
//             {
//                 out[out_indx] = ff_add(out[out_indx], ff_multiply(kernel[in_size - j], signal[i+j-pad_size], PRIME), PRIME);
//             }
//         }
//     }
// }

__device__ void convolve_gpu(const u32 *kernel, const u32 *signal, u32 *out, int kernel_size, int signal_size, int out_start_loc)
{
    int result_size = signal_size+kernel_size-1;
    int pad_size = kernel_size-1;
    for (int i=0; i<result_size; i++)
    {
        int out_indx = out_start_loc+i;
        out[out_indx] = 0;
        for (int j = 0; j < kernel_size; j++)
        {
            if (i+j >= pad_size && i+j-pad_size < signal_size)
            {
                if (j == 0) {
                    // Account for padding
                    out[out_indx] = ff_add(out[out_indx], ff_multiply(0, signal[i+j-pad_size], PRIME), PRIME);
                } else {
                    out[out_indx] = ff_add(out[out_indx], ff_multiply(kernel[kernel_size - 1 - j], signal[i+j-pad_size], PRIME), PRIME);
                }
            }
        }
    }
}

__global__ void lagrange_convolution(u32 *lagrange, const u32 *lagrange_tmp, int level)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // if (idx == 0)
    // {
    //     printf("level: %i \n", level);
    // }

    int sub_pol_size = pow(2, level) + 1;

    int step_size = pow(2, level+2);
    int start_val_ker = idx*step_size;
    int start_val_sig = idx*step_size + step_size/2;

    int kernel_size = sub_pol_size;
    int signal_size = sub_pol_size;

    // For some ungodly reason instantiating the new pointer without const mutates the lagrange_tmp values!!!???!!!
    const u32 *kernel = lagrange_tmp+start_val_ker;
    const u32 *signal = lagrange_tmp+start_val_sig;
    // if (idx == 0)
    // {
    //     printf("0 level: %i start_val_ker: %i start_val_sig: %i \n", level, start_val_ker, start_val_sig);
    //     // print_vec(kernel, signal_size);
    //     // print_vec(signal, signal_size);
    //     print_vec(lagrange_tmp, 20);
    //     print_vec(lagrange, 20);
    // }
    
    // for (int i=0; i<sub_pol_size; i++)
    // {
    //     kernel[i] = lagrange[start_val_ker+i];
    //     signal[i] = lagrange[start_val_sig+i];
    // }

    if (idx == 0)
    {
        printf("level: %i start_val_ker: %i start_val_sig: %i \n", level, start_val_ker, start_val_sig);
        print_vec(kernel, signal_size);
        print_vec(signal, signal_size);
        print_vec(lagrange_tmp, 20);
        print_vec(lagrange, 20);
    }

    // kernel[sub_pol_size] = 0; // Padding for convolution this is causing an issue because the lagrange and tmp arrays become a diffeernet length

    int n_samps = pow(2, 6) + 1;
    int probe_len = pow(n_samps, 1);
    int lagrange_size = (n_samps-1)*n_samps*2;

    if (start_val_ker > lagrange_size)
    {
        printf("error \n");
        return;
    }
    if (start_val_ker < lagrange_size)
    {
        convolve_gpu(kernel, signal, lagrange, kernel_size, signal_size, start_val_ker);
    }

    if (idx == 0)
    {
        print_vec(lagrange, 20);
    }
}

__global__ void init_lagrange_branch_a(const u32* xs, u32* lagrange, u32* lagrange_tmp, int n_samps)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int sub_pol_size = 2;
    int start_index = idx*sub_pol_size;

    int read_index = idx%(n_samps-1);
    if (read_index < idx/(n_samps-1)) 
    {
        lagrange[start_index] = ff_subtract(0, xs[read_index], PRIME);
        lagrange[start_index+1] = 1;

        lagrange_tmp[start_index] = ff_subtract(0, xs[read_index], PRIME);
        lagrange_tmp[start_index+1] = 1;
    }
}

__global__ void init_lagrange_branch_b(const u32* xs, u32* lagrange, u32* lagrange_tmp, int n_samps)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int sub_pol_size = 2;
    int start_index = idx*sub_pol_size;

    int read_index = idx%(n_samps-1);
    if (read_index >= idx/(n_samps-1)) 
    {
        lagrange[start_index] = ff_subtract(0, xs[read_index + 1], PRIME);
        lagrange[start_index+1] = 1;

        lagrange_tmp[start_index] = ff_subtract(0, xs[read_index + 1], PRIME);
        lagrange_tmp[start_index+1] = 1;
    }
}

void multi_interp(int n_vars, int two_exponent)
{
    int n_samps = pow(2, two_exponent) + 1;
    int probe_len = pow(n_samps, n_vars);
    int lagrange_size = n_vars*(n_samps-1)*n_samps*2;
    u32 lagrange_polynomials[lagrange_size];

    u32 probes[probe_len];
    u32 xs[n_vars*n_samps];
    std::srand(time(0));

    for (int i=0; i<n_vars; i++)
    {
        for (int j=0; j<n_samps; j++)
        {
            int flat_index = i*n_samps + j;
            xs[flat_index] = (j+1)%PRIME;
            // xs[flat_index] = (std::rand())%PRIME;
        }
    }

    u32 *d_xs, *d_probes, *d_probes_2, *d_lagrange, *d_lagrange_tmp;

    // Size in bytes for each vector
    size_t bytes_xs = n_vars*n_samps * sizeof(u32);
    size_t bytes_probes = probe_len * sizeof(u32);
    size_t bytes_lagrange = lagrange_size * sizeof(u32);

    CUDA_SAFE_CALL(hipSetDevice(1));

    // Allocate memory on the device
    CUDA_SAFE_CALL(hipMalloc(&d_xs, bytes_xs));
    CUDA_SAFE_CALL(hipMalloc(&d_probes, bytes_probes));
    CUDA_SAFE_CALL(hipMalloc(&d_probes_2, bytes_probes));
    CUDA_SAFE_CALL(hipMalloc(&d_lagrange, bytes_lagrange));
    CUDA_SAFE_CALL(hipMalloc(&d_lagrange_tmp, bytes_lagrange));

    CUDA_SAFE_CALL(hipMemcpy(d_xs, xs, bytes_xs, hipMemcpyHostToDevice));

    int required_threads = probe_len;
    int threadsPerBlock = required_threads>256? 256 : probe_len;
    int blocksPerGrid = (required_threads + threadsPerBlock - 1) / threadsPerBlock;

    // Computre all probes
    compute_probes<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_probes, d_probes_2, n_vars, n_samps);

    required_threads = lagrange_size;
    threadsPerBlock = required_threads>256? 256 : probe_len;
    blocksPerGrid = (required_threads + threadsPerBlock - 1) / threadsPerBlock;

    // Dispatch together TODO
    init_lagrange_branch_a<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_lagrange, d_lagrange_tmp, n_samps);
    init_lagrange_branch_b<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_lagrange, d_lagrange_tmp, n_samps);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    for (int i=0; i<two_exponent; i++)
    {
        required_threads = pow(2, two_exponent-1-i);
        // required_threads = lagrange_size/pow(2, i+2);
        printf("required_threads: %i \n", required_threads);
        threadsPerBlock = required_threads>256? 256 : probe_len;
        blocksPerGrid = (required_threads + threadsPerBlock - 1) / threadsPerBlock;

        lagrange_convolution<<<blocksPerGrid, threadsPerBlock>>>(d_lagrange, d_lagrange_tmp, i);
        std::swap(d_lagrange, d_lagrange_tmp);

        // CUDA_SAFE_CALL(hipDeviceSynchronize());
    }
    std::swap(d_lagrange, d_lagrange_tmp);

    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(lagrange_polynomials, d_lagrange_tmp, bytes_lagrange, hipMemcpyDeviceToHost));



    // Perform multidimensional interpolation
    required_threads = probe_len;
    threadsPerBlock = required_threads>256? 256 : probe_len;
    blocksPerGrid = (required_threads + threadsPerBlock - 1) / threadsPerBlock;
    for (int i=0; i<n_vars; i++)
    {
        get_lagrange_coeffs_nd<<<blocksPerGrid, threadsPerBlock>>>(d_xs, d_probes, d_probes_2, d_lagrange, i, n_vars, n_samps, two_exponent);
        std::swap(d_probes, d_probes_2);
        // CUDA_SAFE_CALL(hipDeviceSynchronize());

    }

    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(probes, d_probes, bytes_probes, hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(lagrange_polynomials, d_lagrange, bytes_lagrange, hipMemcpyDeviceToHost));

    std::vector<double> probe_vec(probe_len);
    for (int i=0; i<probe_len; i++)
    {
        std::cout << "probe: " << probes[i] << " ";
        probe_vec[i] = probes[i];
    }

    for (int i=0; i<lagrange_size; i++)
    {
        if (i%(2*(n_samps-1)) == 0) {
            std::cout << std::endl;
        }
        std::cout << as_int(lagrange_polynomials[i]) << " ";

    }

    std::vector<std::string> vars = {"x", "y", "z"};
    std::string poly = nd_poly_to_string_flat(probe_vec, vars, n_samps);
    std::cout << std::endl << poly << std::endl;

    // Free memory on the device
    CUDA_SAFE_CALL(hipFree(d_xs));
    CUDA_SAFE_CALL(hipFree(d_probes));
    CUDA_SAFE_CALL(hipFree(d_probes_2));
    CUDA_SAFE_CALL(hipFree(d_lagrange));
    CUDA_SAFE_CALL(hipFree(d_lagrange_tmp));

}

// int main()
// {
//     multi_interp(3, 6);
//     return 0;
// }