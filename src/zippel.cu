#include "hip/hip_runtime.h"
#include "GPUFR/zippel.cuh"

#include "GPUFR/ff_math.cuh"

__global__ void evaluate_monomials(u32 n_variables, size_t pitch, u32 *anchor_points, u32 *exponents, u32 *result, u32 p){
	
	i64 idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	u32 r = 1;

	for(size_t i = 0; i < n_variables; i++){
		r = ff_multiply(r, ff_pow(anchor_points[i], exponents[idx + i*pitch], p), p);	
	}

	result[idx] = r;

}
 
__global__ void evaluate_powers(u32 n_variables, u32 max_power, u32 *anchor_points, u32 *result, u32 p){
	
	// x-direction corresponds to different powers
	u32 idx = threadIdx.x + blockIdx.x * blockDim.x;
	// y-direction corresponds to different variables
	u32 idy = threadIdx.y + blockIdx.y * blockDim.y;

	if(idx < max_power && idy < n_variables){
		// Lowest power is 1
		u32 power = idx + 1;
		result[idx + max_power*idy] = ff_pow(anchor_points[idy], power, p);
	}
}
