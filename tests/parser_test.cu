#include "GPUFR/parser.hpp"
#include "GPUFR/types.h"

#include <array>
#include <vector>
#include <deque>
#include <algorithm>
#include <random>
#include <sstream>

#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <flint/flint.h>
#include <flint/nmod_types.h>
#include <flint/nmod.h>

#include <catch2/catch_test_macros.hpp>
#include <catch2/generators/catch_generators_adapters.hpp>
#include <catch2/generators/catch_generators_random.hpp>

TEST_CASE("Parser Test", "[Parsing][Evaluation]"){

	SECTION("Tokenization should reproduce initial string"){
		const std::string expression = "3+x^2+7*x^3";
		
		std::deque<std::string> tokens = tokenize(expression);

		std::stringstream result;
		for(auto token: tokens){
			result << token;
		}

		REQUIRE(result.str() == expression);

	}

	const std::vector<std::string> vars = {"x"};
	const std::string expression = "3 + x^2 + 7*x^3";
	auto black_box = [](u32 x, nmod_t mod){
		u32 result = 3;
		result = nmod_add(result, nmod_pow_ui(x, 2, mod), mod);
		u32 temp = nmod_mul(7, nmod_pow_ui(x, 3, mod), mod);
		result = nmod_add(result, temp, mod);
		return result;
	};

}
