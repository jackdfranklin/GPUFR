#include "hip/hip_runtime.h"
#include "GPUFR/ntt.cuh"

#include <array>
#include <vector>
#include <algorithm>
#include <random>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <catch2/catch_test_macros.hpp>
#include <catch2/generators/catch_generators_adapters.hpp>
#include <catch2/generators/catch_generators_random.hpp>

int as_int(u32 val, u32 prime)
{
    int result = val;
    if (result > prime/2) result = result - prime;
    return result;
}

void print_vec(const u32* vec, int size, u32 prime)
{
    for (int i=0; i<size; i++)
    {
        printf("%i, ", vec[i]);
    }
    printf("\n");
}

TEST_CASE("ntt_test"){
    int deviceCount = 0;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));

    int arr_size = 4;

    u32* in_arr = new u32[arr_size];
    u32* out_arr = new u32[arr_size];

    for (int i=0; i<arr_size; i++)
    {
        in_arr[i] = i+1;
    }

    u32* d_in_arr;
    u32* d_out_arr;

    int bytes_arr = arr_size*sizeof(u32);
    CUDA_SAFE_CALL(hipMalloc(&d_in_arr, bytes_arr));
    CUDA_SAFE_CALL(hipMalloc(&d_out_arr, bytes_arr));

    CUDA_SAFE_CALL(hipMemcpy(d_in_arr, in_arr, bytes_arr, hipMemcpyHostToDevice));

    std::vector<u32> ws = get_w("./precomp/primes_roots_13.csv", 0);
    u32 prime = ws[0];
    do_ntt(d_in_arr, d_out_arr, arr_size, ws, prime);

    CUDA_SAFE_CALL(hipMemcpy(out_arr, d_out_arr, bytes_arr, hipMemcpyDeviceToHost));

    print_vec(out_arr, arr_size, prime);

    CUDA_SAFE_CALL(hipFree(d_in_arr));
    CUDA_SAFE_CALL(hipFree(d_out_arr));

    delete[] in_arr;
    delete[] out_arr;

    REQUIRE(true);
}

TEST_CASE("ntt_and_intt"){
    int deviceCount = 0;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));

    int arr_size = 1<<8;

    u32* in_arr = new u32[arr_size];
    u32* out_arr = new u32[arr_size];

    for (int i=0; i<arr_size; i++)
    {
        in_arr[i] = i+1;
    }

    u32* d_in_arr;
    u32* d_out_arr;

    int bytes_arr = arr_size*sizeof(u32);
    CUDA_SAFE_CALL(hipMalloc(&d_in_arr, bytes_arr));
    CUDA_SAFE_CALL(hipMalloc(&d_out_arr, bytes_arr));

    CUDA_SAFE_CALL(hipMemcpy(d_in_arr, in_arr, bytes_arr, hipMemcpyHostToDevice));

    std::vector<u32> ws = get_w("./precomp/primes_roots_13.csv", 0);
    u32 prime = ws[0];
    do_ntt(d_in_arr, d_out_arr, arr_size, ws, prime);
    do_ntt(d_out_arr, d_in_arr, arr_size, ws, prime, true);

    CUDA_SAFE_CALL(hipMemcpy(out_arr, d_in_arr, bytes_arr, hipMemcpyDeviceToHost));

    for (int i=0; i<arr_size; i++)
    {
        REQUIRE(in_arr[i] == out_arr[i]);
    }

    CUDA_SAFE_CALL(hipFree(d_in_arr));
    CUDA_SAFE_CALL(hipFree(d_out_arr));

    delete[] in_arr;
    delete[] out_arr;
}

TEST_CASE("ntt_bulk"){
    int deviceCount = 0;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));

    int exp = 2;
    int n_samps = (1<<exp) + 1;
    int initial_pol_size = 4;
    int arr_size = n_samps*(n_samps-1)*initial_pol_size;

    u32* in_arr = new u32[arr_size];
    u32* out_arr = new u32[arr_size];

    // for (int i=0; i<arr_size; i++)
    // {
    //     if (i%initial_pol_size == 0 || i%initial_pol_size == 1)
    //     {
    //         in_arr[i] = 0;
    //     } else if (i%initial_pol_size == 2)
    //     {
    //         in_arr[i] = 1;
    //     } else {
    //         in_arr[i] = i+1;
    //     }
    // }

    for (int i=0; i<arr_size; i++)
    {
        in_arr[i] = i%initial_pol_size + 1;
    }

    u32* d_in_arr;
    u32* d_out_arr;

    int bytes_arr = arr_size*sizeof(u32);
    CUDA_SAFE_CALL(hipMalloc(&d_in_arr, bytes_arr));
    CUDA_SAFE_CALL(hipMalloc(&d_out_arr, bytes_arr));

    CUDA_SAFE_CALL(hipMemcpy(d_in_arr, in_arr, bytes_arr, hipMemcpyHostToDevice));

    std::vector<u32> ws = get_w("./precomp/primes_roots_13.csv", 0);
    u32 prime = ws[0];
    do_bulk_ntt(d_in_arr, d_out_arr, n_samps, 0, ws, prime);
    do_bulk_ntt(d_out_arr, d_in_arr, n_samps, 0, ws, prime, true);

    CUDA_SAFE_CALL(hipMemcpy(out_arr, d_in_arr, bytes_arr, hipMemcpyDeviceToHost));

    // print_vec(out_arr, arr_size, prime);

    for (int i=0; i<arr_size; i++)
    {
        REQUIRE(in_arr[i] == out_arr[i]);
    }

    CUDA_SAFE_CALL(hipFree(d_in_arr));
    CUDA_SAFE_CALL(hipFree(d_out_arr));

    delete[] in_arr;
    delete[] out_arr;
}