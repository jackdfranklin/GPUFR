#include "hip/hip_runtime.h"
#include "GPUFR/ff_math.cuh"
#include "GPUFR/types.h"

#include <array>
#include <vector>
#include <algorithm>
#include <random>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <flint/flint.h>
#include <flint/nmod_types.h>
#include <flint/nmod.h>

#include <catch2/catch_test_macros.hpp>

__global__ void ff_add_test(u32 p, u32 *a, u32 *b, u32 *c){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	c[idx] = ff_add(a[idx], b[idx], p);	
}

__global__ void ff_subtract_test(u32 p, u32 *a, u32 *b, u32 *c){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	c[idx] = ff_subtract(a[idx], b[idx], p);	
}

__global__ void ff_multiply_test(u32 p, u32 *a, u32 *b, u32 *c){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	c[idx] = ff_multiply(a[idx], b[idx], p);	
}

__global__ void ff_inverse_test(u32 p, u32 *a, u32 *c){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(a[idx]%p != 0){
		u32 a_inv = modular_inverse(a[idx], p);
		c[idx] = ff_multiply(a[idx], a_inv, p);
	}
	else {
		c[idx] = 1;
	}
}

__global__ void ff_divide_test(u32 p, u32 *a, u32 *b, u32 *c){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	c[idx] = ff_divide(a[idx], b[idx], p);	
}

TEST_CASE("First Test", "[Finite Field]"){
	// The setup is repeated before running each section
	std::array<u32, 3> p_array = {13, 2546604103, 3998191247};

	u32 number_of_values = 128;
	std::vector<u32> a(number_of_values), b(number_of_values), c(number_of_values);

	std::random_device rand_device;
	std::mt19937 mersenne_engine {rand_device()};
	std::uniform_int_distribution<u32> dist {0, UINT32_MAX};

	auto generator = [&](){
		return dist(mersenne_engine);
	};

	std::generate(a.begin(), a.end(), generator);
	std::generate(b.begin(), b.end(), generator);

	u32 *d_a, *d_b, *d_c;

	hipMalloc(&d_a, number_of_values*sizeof(u32));
	hipMalloc(&d_b, number_of_values*sizeof(u32));
	hipMalloc(&d_c, number_of_values*sizeof(u32));

	hipMemcpy(d_a, a.data(), number_of_values*sizeof(u32), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), number_of_values*sizeof(u32), hipMemcpyHostToDevice);

	SECTION("Addition"){

		for(u32 p: p_array){
			
			ff_add_test<<<1,128>>>(p, d_a, d_b, d_c);
			hipDeviceSynchronize();
			hipMemcpy(c.data(), d_c, number_of_values*sizeof(u32), hipMemcpyDeviceToHost);

			nmod_t modulus = {0};
			nmod_init(&modulus, p);

			for(size_t i = 0; i < number_of_values; i++){
				INFO("p = "<<p<<", a = "<<a.at(i)<<", b = "<<b.at(i)<<", c = "<<c[i]);
				REQUIRE(c[i] == nmod_add(a.at(i)%p, b.at(i)%p, modulus));
			}

		}
		
	}

	SECTION("Subtraction"){

		for(u32 p: p_array){
			
			ff_subtract_test<<<1,128>>>(p, d_a, d_b, d_c);
			hipDeviceSynchronize();
			hipMemcpy(c.data(), d_c, number_of_values*sizeof(u32), hipMemcpyDeviceToHost);

			nmod_t modulus = {0};
			nmod_init(&modulus, p);

			for(size_t i = 0; i < number_of_values; i++){
				INFO("p = "<<p<<", a = "<<a.at(i)<<", b = "<<b.at(i)<<", c = "<<c[i]);
				REQUIRE(c[i] == nmod_sub(a.at(i)%p, b.at(i)%p, modulus));
			}

		}
		
	}

	SECTION("Multiplication"){

		for(u32 p: p_array){
			
			ff_multiply_test<<<1,128>>>(p, d_a, d_b, d_c);
			hipDeviceSynchronize();
			hipMemcpy(c.data(), d_c, number_of_values*sizeof(u32), hipMemcpyDeviceToHost);

			nmod_t modulus = {0};
			nmod_init(&modulus, p);

			for(size_t i = 0; i < number_of_values; i++){
				INFO("p = "<<p<<", a = "<<a.at(i)<<", b = "<<b.at(i)<<", c = "<<c[i]);
				CHECK(c[i] == nmod_mul(a.at(i)%p, b.at(i)%p, modulus));
			}

		}
		
	}

	SECTION("Multiplicative Inverse"){

		for(u32 p: p_array){

			ff_inverse_test<<<1,128>>>(p, d_a, d_c);
			hipDeviceSynchronize();
			hipMemcpy(c.data(), d_c, number_of_values*sizeof(u32), hipMemcpyDeviceToHost);

			nmod_t modulus = {0};
			nmod_init(&modulus, p);

			for(size_t i = 0; i < number_of_values; i++){
				INFO("p = "<<p<<", a = "<<a.at(i)<<", b = "<<b.at(i)<<", c = "<<c[i]);
				REQUIRE(c[i] == 1);
			}

		}
	}

	SECTION("Division"){

		for(u32 p: p_array){

			std::uniform_int_distribution<u32> dist {1, p-1};

			auto generator = [&](){
				return dist(mersenne_engine);
			};

			std::generate(a.begin(), a.end(), generator);
			std::generate(b.begin(), b.end(), generator);

			hipMemcpy(d_a, a.data(), number_of_values*sizeof(u32), hipMemcpyHostToDevice);
			hipMemcpy(d_b, b.data(), number_of_values*sizeof(u32), hipMemcpyHostToDevice);

			ff_divide_test<<<1,128>>>(p, d_a, d_b, d_c);
			hipDeviceSynchronize();
			hipMemcpy(c.data(), d_c, number_of_values*sizeof(u32), hipMemcpyDeviceToHost);

			nmod_t modulus = {0};
			nmod_init(&modulus, p);

			for(size_t i = 0; i < number_of_values; i++){
				INFO("p = "<<p<<", a = "<<a.at(i)<<", b = "<<b.at(i)<<", c = "<<c[i]);
				REQUIRE(c[i] == nmod_div(a.at(i)%p, b.at(i)%p, modulus));
			}

		}
	}
}
