#include "hip/hip_runtime.h"
#include <catch2/catch_test_macros.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void test_kernel(bool *out_param){
	*out_param = true;
}

TEST_CASE("CUDA Test", "[CUDA]"){
	hipSetDevice(0);

	bool *d_result;
	hipMalloc(&d_result, sizeof(bool));

	test_kernel<<<1,1>>>(d_result);

	bool h_result = false;
	hipMemcpy(&h_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	REQUIRE(h_result == true);	
}

